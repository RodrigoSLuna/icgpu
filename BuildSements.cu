
#include <hip/hip_runtime.h>
#include <cstdio>


const int N = 4;
const int TAM_BLOCO = 2;
#define CUDA_SAFE_CALL(call) { \
cudaError_t err = call ; \
if( err != hipSuccess ) { \
fprintf ( stderr ," Erro no arquivo ’%s ’, linha %i: %s.\n", \
__FILE__ , __LINE__ , hipGetErrorString ( err ) ) ; \
exit ( EXIT_FAILURE ) ;\
}\
}

__global__ void Build(const double *X, const double *Y, const double *Z, double *SEG, double *acc_angle){
	int i = blockIdx.x *blockDim.x + threadIdx.x;
	int j = blockIdx.y *blockDim.y + threadIdx.y;


}


int main(){
	double *h_x, *h_y,*h_z, *h_seg, *h_angle;
	double *d_x, *d_y, *d_z, *d_seg, *d_angle;

	int D2_bytes = N*N;
	int D3_bytes = N*N*N*sizeof(double);

	h_x = (double*)malloc(D2_bytes);
	h_y = (double*)malloc(D2_bytes);
	h_z = (double*)malloc(D3_bytes);
	h_angle = (double*)malloc( sizeof(double) );
	h_seg = (double*)malloc(D3_bytes);

	if(h_x == NULL || h_y == NULL || h_z == NULL || h_seg) exit(EXIT_FAILURE);

	//leitura dos dados que estao no host
	//inicializo d_seg com -1
	memset(h_seg,0,D3_bytes);
	for(int i = 0;i<N;i++){
		for(int j = 0;j<N;j++){
			int pos = N*i+j;
			scanf("%lf %lf %lf",&h_x[pos],&h_y[pos],&h_z[pos]);
		}
	}

	CUDA_SAFE_CALL(	hipMalloc( (void**) &d_x, D2_bytes  ) );
	CUDA_SAFE_CALL( hipMalloc( (void**) &d_y, D2_bytes  ) );
	CUDA_SAFE_CALL( hipMalloc( (void**) &d_z, D2_bytes  ) );
	CUDA_SAFE_CALL( hipMalloc( (void**) &d_seg, D3_bytes  ) );
	CUDA_SAFE_CALL( hipMalloc( (void**) &d_angle, sizeof(double)  ) );


	CUDA_SAFE_CALL(	hipMemcpy(d_x,h_y,D2_bytes,hipMemcpyHostToDevice ));
	CUDA_SAFE_CALL(	hipMemcpy(d_y,h_y,D2_bytes,hipMemcpyHostToDevice ));
	CUDA_SAFE_CALL(	hipMemcpy(d_z,h_z,D2_bytes,hipMemcpyHostToDevice ));
	CUDA_SAFE_CALL(	hipMemcpy(d_seg,h_seg,D3_bytes,hipMemcpyHostToDevice ));
	CUDA_SAFE_CALL(	hipMemcpy(d_angle,h_angle, sizeof( double ),hipMemcpyHostToDevice ));


	// Crio os eventos para poder contabilizar o tempo gasto para o processamento na GPU
	hipEvent_t start, stop;
	CUDA_SAFE_CALL(hipEventCreate(&start));
	CUDA_SAFE_CALL(hipEventCreate(&stop));

	/*
	int n_threads = 1024;
	int n_blocos = (N+n_threads-1)/n_threads;
	*/

	CUDA_SAFE_CALL(hipEventRecord(start));
	

	dim3 threadsBloco(TAM_BLOCO,TAM_BLOCO); // no max 1024 threads por bloco
	dim3 blocosGrade(N/threadsBloco.x, N/threadsBloco.y );
	/*
		Chamada do kernel
	*/
	Build<<<blocosGrade,threadsBloco >>>(d_x,d_y,d_z,d_seg, d_angle);
	CUDA_SAFE_CALL ( hipGetLastError () ) ;
	CUDA_SAFE_CALL(hipEventRecord(start));

	float delta_eventos = 0;
	CUDA_SAFE_CALL(hipEventElapsedTime(&delta_eventos,start,stop));

	CUDA_SAFE_CALL(	hipMemcpy(d_seg,h_seg,D3_bytes,hipMemcpyDeviceToHost ));

	CUDA_SAFE_CALL ( hipFree ( d_x ) ) ;
	CUDA_SAFE_CALL ( hipFree ( d_y ) ) ;
	CUDA_SAFE_CALL ( hipFree ( d_z ) ) ;
	CUDA_SAFE_CALL ( hipFree ( d_seg ) ) ;




	return 0;
}