
#include <hip/hip_runtime.h>
#include <cstdio>


/*
Tutorial.
https://devblogs.nvidia.com/even-easier-introduction-cuda/
*/

int N = 3;
const int TAM_BLOCO = 3;
#define CUDA_SAFE_CALL(call) { \
cudaError_t err = call ; \
if( err != hipSuccess ) { \
fprintf ( stderr ," Erro no arquivo ’%s ’, linha %i: %s.\n", \
__FILE__ , __LINE__ , hipGetErrorString ( err ) ) ; \
exit ( EXIT_FAILURE ) ;\
}\
}


//(2* (*N)) + pos
//TODO
//estou usando muitas instruções 
__global__ void Build(const double *X, const double *Y, const double *Z,double *SEG, int *N  ,double *acc_angle){
	unsigned int i = blockIdx.x *blockDim.x + threadIdx.x; //id sensor
	unsigned int j = blockIdx.y *blockDim.y + threadIdx.y; //position particle j 
	unsigned int k = blockIdx.z *blockDim.z + threadIdx.z; //position particle k 
	int pos = i* (*N) + j;

	if( i+2 >= *N or j> (*N) or k> (*N))
		return;
	double x = X[ (i+2)* (*N) +k  ] - X[ pos ];
	double y = Y[ (i+2)* (*N) +k  ] - Y[ pos ];
	double z = Z[ (i+2)* (*N) +k  ] - Z[ pos ];
	double tx = x/z;
	double ty = y/z;

	unsigned int idx = k* (*N)* (*N) + j* (*N) + i;
	if(tx*tx + ty*ty <= *acc_angle){ 		//levar pra funcao, acc_angle ao quadrado!
		//printf("sensor: i: %d particula j: %d particula k: %d \n",i,j,k);
		SEG[idx] = 1;
	}

}


int main(){
	
	int D2_bytes = N*N;
	int D3_bytes = N*N*N;
	double h_x[D2_bytes],h_y[D2_bytes],h_z[D2_bytes],h_seg[D3_bytes], h_angle = 4; // angle already ^2
	double *d_x, *d_y, *d_z, *d_seg, *d_angle;
	int    *d_N,h_N;



	//leitura dos dados que estao no host
	
	//inicializo d_seg com -1
	memset(h_seg,0,D3_bytes);
	
	scanf("%d",&h_N);
	
	for(int i = 0;i<N;i++){
		for(int j = 0;j<N;j++){
			int pos = N*i+j;
			scanf("%lf",&h_x[pos]);
		}
	}
	for(int i = 0;i<N;i++){
		for(int j = 0;j<N;j++){
			int pos = N*i+j;
			scanf("%lf",&h_y[pos]);
		}
	}
	for(int i = 0;i<N;i++){
		for(int j = 0;j<N;j++){
			int pos = N*i+j;
			scanf("%lf",&h_z[pos]);
		}
	}

	CUDA_SAFE_CALL(	hipMalloc( (void**) &d_N, sizeof(int)  ) );
	CUDA_SAFE_CALL(	hipMalloc( (void**) &d_x, D2_bytes*sizeof(double)  ) );
	CUDA_SAFE_CALL( hipMalloc( (void**) &d_y, D2_bytes*sizeof(double)  ) );
	CUDA_SAFE_CALL( hipMalloc( (void**) &d_z, D2_bytes*sizeof(double)  ) );
	CUDA_SAFE_CALL( hipMalloc( (void**) &d_seg, D3_bytes*sizeof(double)  ) );
	CUDA_SAFE_CALL( hipMalloc( (void**) &d_angle, sizeof(double)  ) );

	CUDA_SAFE_CALL(	hipMemcpy(d_N,&h_N  , sizeof(int)	,hipMemcpyHostToDevice ));
	CUDA_SAFE_CALL(	hipMemcpy(d_x,&h_y,   D2_bytes*sizeof(double)		,hipMemcpyHostToDevice ));
	CUDA_SAFE_CALL(	hipMemcpy(d_y,&h_y,D2_bytes*sizeof(double)		,hipMemcpyHostToDevice ));
	CUDA_SAFE_CALL(	hipMemcpy(d_z,&h_z,D2_bytes*sizeof(double)		,hipMemcpyHostToDevice ));
	CUDA_SAFE_CALL(	hipMemcpy(d_seg,&h_seg,D3_bytes*sizeof(double)	,hipMemcpyHostToDevice ));
	CUDA_SAFE_CALL(	hipMemcpy(d_angle,&h_angle, sizeof( double ),hipMemcpyHostToDevice ));

	// Crio os eventos para poder contabilizar o tempo gasto para o processamento na GPU
	hipEvent_t start, stop;
	CUDA_SAFE_CALL(hipEventCreate(&start));
	CUDA_SAFE_CALL(hipEventCreate(&stop));

	/*
	int n_threads = 1024;
	int n_blocos = (N+n_threads-1)/n_threads;
	*/

	CUDA_SAFE_CALL(hipEventRecord(start));
	

	dim3 threadsBloco(TAM_BLOCO,TAM_BLOCO,TAM_BLOCO); // 4 threads nesse bloco
	dim3 blocosGrade(N/threadsBloco.x, N/threadsBloco.y,N/threadsBloco.z ); // quantidade de blocos
	/*
		Chamada do kernel
	*/
	Build<<<blocosGrade,threadsBloco >>>(d_x,d_y,d_z,d_seg,d_N, d_angle);
	CUDA_SAFE_CALL ( hipGetLastError () ) ;
	//
	CUDA_SAFE_CALL(hipEventRecord(stop));
	CUDA_SAFE_CALL ( hipEventSynchronize ( stop ) );

	float delta_eventos = 0;


	CUDA_SAFE_CALL(hipEventElapsedTime(&delta_eventos,start,stop));
	printf("Tempo: %lf \n",delta_eventos );
	CUDA_SAFE_CALL(	hipMemcpy(&h_seg,d_seg,D3_bytes*sizeof(double),hipMemcpyDeviceToHost ));
	
	for(int i =0 ;i<D3_bytes;i++)
		printf("%lf\n",h_seg[i]);
	
	CUDA_SAFE_CALL ( hipFree ( d_x ) ) ;
	CUDA_SAFE_CALL ( hipFree ( d_y ) ) ;
	CUDA_SAFE_CALL ( hipFree ( d_z ) ) ;
	CUDA_SAFE_CALL ( hipFree ( d_seg ) ) ;




	return 0;
}