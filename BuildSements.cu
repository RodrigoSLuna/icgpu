#include "hip/hip_runtime.h"
#include <cstdio>


/*
Tutorial.
https://devblogs.nvidia.com/even-easier-introduction-cuda/
*/

int N = 4;
const int TAM_BLOCO = 2;
#define CUDA_SAFE_CALL(call) { \
hipError_t err = call ; \
if( err != hipSuccess ) { \
fprintf ( stderr ," Erro no arquivo ’%s ’, linha %i: %s.\n", \
__FILE__ , __LINE__ , hipGetErrorString ( err ) ) ; \
exit ( EXIT_FAILURE ) ;\
}\
}



//TODO
//estou usando muitas instruções 
__global__ void Build(const double *X, const double *Y, const double *Z,double *SEG, unsigned int N  ,double acc_angle){
	unsigned int i = blockIdx.x *blockDim.x + threadIdx.x; //id sensor
	unsigned int j = blockIdx.y *blockDim.y + threadIdx.y; //positions particle  
	unsigned int k = blockIdx.z *blockDim.z + threadIdx.z;

	int pos = i*N + j;
	if(i>N or j>N or k>N)
		return;

	double x = X[ (2*N) + pos ] - X[ pos ];
	double y = Y[ (2*N) + pos ] - Y[ pos ];
	double z = Z[ (2*N) + pos ] - Z[ pos ];
	double tx = x/z;
	double ty = y/z;

	unsigned int idx = k*N*N + j*N + i; 	 // levar pra funcao N^2 e N.

	if(tx*tx + ty*ty <= acc_angle*acc_angle) //levar pra funcao, acc_angle ao quadrado!
		SEG[idx] = 1;

}


int main(){
	double *h_x, *h_y,*h_z, *h_seg, *h_angle,*h_N;
	double *d_x, *d_y, *d_z, *d_seg, *d_angle;
	int    *d_N;

	int D2_bytes = N*N;
	int D3_bytes = N*N*N*sizeof(double);



	h_x = (double*)malloc(D2_bytes);
	h_x = (double*)malloc(D2_bytes);
	h_y = (double*)malloc(D2_bytes);
	h_z = (double*)malloc(D3_bytes);
	h_angle = (double*)malloc( sizeof(double) );
	h_seg = (double*)malloc(D3_bytes);

	if(h_x == NULL || h_y == NULL || h_z == NULL || h_seg) exit(EXIT_FAILURE);

	//leitura dos dados que estao no host
	//inicializo d_seg com -1
	memset(h_seg,0,D3_bytes);
	for(int i = 0;i<N;i++){
		for(int j = 0;j<N;j++){
			int pos = N*i+j;
			scanf("%lf %lf %lf",&h_x[pos],&h_y[pos],&h_z[pos]);
		}
	}

	CUDA_SAFE_CALL(	hipMalloc( (void**) &d_N, sizeof(int)  ) );
	CUDA_SAFE_CALL(	hipMalloc( (void**) &d_x, D2_bytes  ) );
	CUDA_SAFE_CALL( hipMalloc( (void**) &d_y, D2_bytes  ) );
	CUDA_SAFE_CALL( hipMalloc( (void**) &d_z, D2_bytes  ) );
	CUDA_SAFE_CALL( hipMalloc( (void**) &d_seg, D3_bytes  ) );
	CUDA_SAFE_CALL( hipMalloc( (void**) &d_angle, sizeof(double)  ) );

	CUDA_SAFE_CALL(	hipMemcpy(d_N,N  , sizeof(int)	,hipMemcpyHostToDevice ));
	CUDA_SAFE_CALL(	hipMemcpy(d_x,h_y,D2_bytes		,hipMemcpyHostToDevice ));
	CUDA_SAFE_CALL(	hipMemcpy(d_y,h_y,D2_bytes		,hipMemcpyHostToDevice ));
	CUDA_SAFE_CALL(	hipMemcpy(d_z,h_z,D2_bytes		,hipMemcpyHostToDevice ));
	CUDA_SAFE_CALL(	hipMemcpy(d_seg,h_seg,D3_bytes	,hipMemcpyHostToDevice ));
	CUDA_SAFE_CALL(	hipMemcpy(d_angle,h_angle, sizeof( double ),hipMemcpyHostToDevice ));


	// Crio os eventos para poder contabilizar o tempo gasto para o processamento na GPU
	hipEvent_t start, stop;
	CUDA_SAFE_CALL(hipEventCreate(&start));
	CUDA_SAFE_CALL(hipEventCreate(&stop));

	/*
	int n_threads = 1024;
	int n_blocos = (N+n_threads-1)/n_threads;
	*/

	CUDA_SAFE_CALL(hipEventRecord(start));
	

	dim3 threadsBloco(TAM_BLOCO,TAM_BLOCO); // 4 threads nesse bloco
	dim3 blocosGrade(N/threadsBloco.x, N/threadsBloco.y ); // quantidade de blocos
	/*
		Chamada do kernel
	*/
	Build<<<blocosGrade,threadsBloco >>>(d_x,d_y,d_z,d_seg, d_angle, d_N);
	CUDA_SAFE_CALL ( hipGetLastError () ) ;
	CUDA_SAFE_CALL(hipEventRecord(start));

	float delta_eventos = 0;
	CUDA_SAFE_CALL(hipEventElapsedTime(&delta_eventos,start,stop));

	CUDA_SAFE_CALL(	hipMemcpy(d_seg,h_seg,D3_bytes,hipMemcpyDeviceToHost ));

	CUDA_SAFE_CALL ( hipFree ( d_x ) ) ;
	CUDA_SAFE_CALL ( hipFree ( d_y ) ) ;
	CUDA_SAFE_CALL ( hipFree ( d_z ) ) ;
	CUDA_SAFE_CALL ( hipFree ( d_seg ) ) ;




	return 0;
}