#include "hip/hip_runtime.h"
#include "MakeSegmentParal.h"


//Mapping
//Flat[x + WIDTH * (y + DEPTH * z)] = Original[x, y, z]
/*
Flat 3D -> 1D
z = idx / (xMax * yMax);
idx -= (z * xMax * yMax);
int y = idx / xMax;
int x = idx % xMax;
*/

// Confirmar cubo angulo


//TODO
//Colocar parametros de entrada, nro de threads, dimensao..

// TODO OK
// Conferir mapeamento das threads, proxima localidade.
// particle i
// particle j
// sensor k
// Ter 3 cubos, para cubo, tx, ty.
 



__global__ void Build(double *TX, double *TY ,const double *X, const double *Y, const double *Z,double *SEG, int N  ,double acc_angle){
	unsigned int i = blockIdx.x *blockDim.x + threadIdx.x; //particle i
	unsigned int j = blockIdx.y *blockDim.y + threadIdx.y; //particle j 
	unsigned int k = blockIdx.z *blockDim.z + threadIdx.z; //sensor k 
	int pos = k* (N) + i;

	double x = X[ (k+2)* (N) +j  ] - X[ pos ];
	double y = Y[ (k+2)* (N) +j  ] - Y[ pos ];
	double z = Z[ (k+2)* (N) +j  ] - Z[ pos ];
	double tx = x/z;
	double ty = y/z;

	unsigned int idx = k* (N * N) + i*(N) + j;
	double b_angle = tx*tx + ty*ty;
	if( b_angle <= (acc_angle) * (acc_angle)){	
		SEG[idx] = b_angle;
		TX[idx] = tx;
		TY[idx] = ty;
	}
	else{
		TX[idx]  = -1;
		TY[idx]  = -1;
		SEG[idx] = -1; // -1 ou 0?
	}
}


double HostBuild(double M_x[64][64], double M_y[64][64],double M_z[64][64],double M_seg[64][64][64],double d_angle){

	int D2_bytes = 64*64;
	int D3_bytes = 64*64*64;
	double h_x[D2_bytes],h_y[D2_bytes],h_z[D2_bytes],h_seg[D3_bytes]; 
	double *d_x, *d_y, *d_z, *d_seg,*d_TX,*d_TY;
	int     d_N = 64;



	//leitura dos dados que estao no host
	
	//inicializo d_seg com -1
	memset(h_seg,0,D3_bytes);
		
	for(int i = 0;i<64;i++){
		for(int j = 0;j<64;j++){
			int pos = 64*i+j;
			h_x[pos] = M_x[i][j];
			h_y[pos] = M_y[i][j];
			h_z[pos] = M_z[i][j];
		}
	}
	
	CUDA_SAFE_CALL(	hipMalloc( (void**) &d_x 		, D2_bytes*sizeof(double)  ) );
	CUDA_SAFE_CALL( hipMalloc( (void**) &d_y 		, D2_bytes*sizeof(double)  ) );
	CUDA_SAFE_CALL( hipMalloc( (void**) &d_z 		, D2_bytes*sizeof(double)  ) );
	CUDA_SAFE_CALL( hipMalloc( (void**) &d_seg 	, D3_bytes*sizeof(double)  ) );
	CUDA_SAFE_CALL( hipMalloc( (void**) &d_TX 		, D3_bytes*sizeof(double)  ) );
	CUDA_SAFE_CALL( hipMalloc( (void**) &d_TY 		, D3_bytes*sizeof(double)  ) );


	CUDA_SAFE_CALL(	hipMemcpy(d_x,&h_x,  D2_bytes*sizeof(double)		,hipMemcpyHostToDevice ));
	CUDA_SAFE_CALL(	hipMemcpy(d_y,&h_y,  D2_bytes*sizeof(double)		,hipMemcpyHostToDevice ));
	CUDA_SAFE_CALL(	hipMemcpy(d_z,&h_z,  D2_bytes*sizeof(double)		,hipMemcpyHostToDevice ));
	CUDA_SAFE_CALL(	hipMemcpy(d_seg,&h_seg,D3_bytes*sizeof(double)		,hipMemcpyHostToDevice ));

	// Crio os eventos para poder contabilizar o tempo gasto para o processamento na GPU
	hipEvent_t start, stop;
	CUDA_SAFE_CALL(hipEventCreate(&start));
	CUDA_SAFE_CALL(hipEventCreate(&stop));

	//max 16.
	int n_threads_dim = 8;
	CUDA_SAFE_CALL(hipEventRecord(start));
	

	dim3 threadsBloco(n_threads_dim,n_threads_dim,n_threads_dim);
	dim3 blocosGrade( N/threadsBloco.x, N/threadsBloco.y,N/threadsBloco.z ); // quantidade de blocos
	//
	//	Chamada do kernel
	//
	Build<<<blocosGrade,threadsBloco >>>(d_TX,d_TY,d_x,d_y,d_z,d_seg,d_N, d_angle);
	CUDA_SAFE_CALL ( hipGetLastError () ) ;
	//
	CUDA_SAFE_CALL ( hipEventRecord(stop)    );
	CUDA_SAFE_CALL ( hipEventSynchronize ( stop ) );

	float delta_eventos = 0;


	CUDA_SAFE_CALL(hipEventElapsedTime(&delta_eventos,start,stop));
//	printf("Tempo em seg: %lf \n",delta_eventos/1000 );
	CUDA_SAFE_CALL(	hipMemcpy(&h_seg,d_seg,D3_bytes*sizeof(double),hipMemcpyDeviceToHost ));
	
	CUDA_SAFE_CALL(	hipMemcpy(&h_seg,d_seg,D3_bytes*sizeof(double)	,hipMemcpyDeviceToHost ));

	for(int i = 0;i<64;i++){
		for(int j = 0;j<64;j++){
			for(int k = 0;k<64;k++){
				int pos = k*64*64 + j*64 +i;
				M_seg[i][j][k] = h_seg[pos];
			}
		}
	}



	CUDA_SAFE_CALL ( hipFree ( d_x ) ) ;
	CUDA_SAFE_CALL ( hipFree ( d_y ) ) ;
	CUDA_SAFE_CALL ( hipFree ( d_z ) ) ;
	CUDA_SAFE_CALL ( hipFree ( d_seg ) ) ;
	return delta_eventos/1000;
}

/*
int main(){
	
	int D2_bytes = N*N;
	int D3_bytes = N*N*N;
	double h_x[D2_bytes],h_y[D2_bytes],h_z[D2_bytes],h_seg[D3_bytes], h_angle = 2; 
	double *d_x, *d_y, *d_z, *d_seg, d_angle;
	int    *d_N, h_N;



	//leitura dos dados que estao no host
	
	//inicializo d_seg com -1
	memset(h_seg,0,D3_bytes);
	
	scanf("%d",&h_N);
	
	for(int i = 0;i<N;i++){
		for(int j = 0;j<N;j++){
			int pos = N*i+j;
			scanf("%lf",&h_x[pos]);
		}
	}
	for(int i = 0;i<N;i++){
		for(int j = 0;j<N;j++){
			int pos = N*i+j;
			scanf("%lf",&h_y[pos]);
		}
	}
	for(int i = 0;i<N;i++){
		for(int j = 0;j<N;j++){
			int pos = N*i+j;
			scanf("%lf",&h_z[pos]);
		}
	}

	CUDA_SAFE_CALL(	hipMalloc( (void**) &d_N, sizeof(int)  ) );
	CUDA_SAFE_CALL(	hipMalloc( (void**) &d_x, D2_bytes*sizeof(double)  ) );
	CUDA_SAFE_CALL( hipMalloc( (void**) &d_y, D2_bytes*sizeof(double)  ) );
	CUDA_SAFE_CALL( hipMalloc( (void**) &d_z, D2_bytes*sizeof(double)  ) );
	CUDA_SAFE_CALL( hipMalloc( (void**) &d_seg, D3_bytes*sizeof(double)  ) );
	CUDA_SAFE_CALL( hipMalloc( (void**) &d_angle, sizeof(double)  ) );

	CUDA_SAFE_CALL(	hipMemcpy(d_N,&h_N  , sizeof(int)	,hipMemcpyHostToDevice ));
	CUDA_SAFE_CALL(	hipMemcpy(d_x,&h_y,   D2_bytes*sizeof(double)		,hipMemcpyHostToDevice ));
	CUDA_SAFE_CALL(	hipMemcpy(d_y,&h_y,D2_bytes*sizeof(double)		,hipMemcpyHostToDevice ));
	CUDA_SAFE_CALL(	hipMemcpy(d_z,&h_z,D2_bytes*sizeof(double)		,hipMemcpyHostToDevice ));
	CUDA_SAFE_CALL(	hipMemcpy(d_seg,&h_seg,D3_bytes*sizeof(double)	,hipMemcpyHostToDevice ));
	CUDA_SAFE_CALL(	hipMemcpy(d_angle,&h_angle, sizeof( double ),hipMemcpyHostToDevice ));

	// Crio os eventos para poder contabilizar o tempo gasto para o processamento na GPU
	hipEvent_t start, stop;
	CUDA_SAFE_CALL(hipEventCreate(&start));
	CUDA_SAFE_CALL(hipEventCreate(&stop));


	int n_threads = 1024;
	int n_blocos = (N+n_threads-1)/n_threads;


	CUDA_SAFE_CALL(hipEventRecord(start));
	

	dim3 threadsBloco(TAM_BLOCO,TAM_BLOCO,TAM_BLOCO); // 4 threads nesse bloco
	dim3 blocosGrade(N/threadsBloco.x, N/threadsBloco.y,N/threadsBloco.z ); // quantidade de blocos
	//
		Chamada do kernel
	//
	Build<<<blocosGrade,threadsBloco >>>(d_x,d_y,d_z,d_seg,d_N, d_angle);
	CUDA_SAFE_CALL ( hipGetLastError () ) ;
	//
	CUDA_SAFE_CALL(hipEventRecord(stop));
	CUDA_SAFE_CALL ( hipEventSynchronize ( stop ) );

	float delta_eventos = 0;


	CUDA_SAFE_CALL(hipEventElapsedTime(&delta_eventos,start,stop));
	printf("Tempo: %lf \n",delta_eventos );
	CUDA_SAFE_CALL(	hipMemcpy(&h_seg,d_seg,D3_bytes*sizeof(double),hipMemcpyDeviceToHost ));
	
	for(int i =0 ;i<D3_bytes;i++)
		printf("%lf\n",h_seg[i]);
	
	CUDA_SAFE_CALL ( hipFree ( d_x ) ) ;
	CUDA_SAFE_CALL ( hipFree ( d_y ) ) ;
	CUDA_SAFE_CALL ( hipFree ( d_z ) ) ;
	CUDA_SAFE_CALL ( hipFree ( d_seg ) ) ;




	return 0;
}
*/